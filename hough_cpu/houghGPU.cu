#include "hip/hip_runtime.h"
// Imported from https://rosettacode.org/wiki/Example:Hough_transform/C
// It will be used as a baseline to observe transformation
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
 
#include "cairo.h"
 
#ifndef M_PI
#define M_PI 3.1415927
#endif
 
#define GR(X,Y) (d[(*s)*(Y)+bpp*(X)+((2)%bpp)])
#define GG(X,Y) (d[(*s)*(Y)+bpp*(X)+((1)%bpp)])
#define GB(X,Y) (d[(*s)*(Y)+bpp*(X)+((0)%bpp)])
#define SR(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+2])
#define SG(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+1])
#define SB(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+0])
#define RAD(A)  (M_PI*((double)(A))/180.0)
uint8_t *houghtransform(uint8_t *d, int *w, int *h, int *s, int bpp)
{
    int rho, theta, y, x, W = *w, H = *h;
    int th = sqrt(W*W + H*H)/2.0;
    int tw = 360;
    uint8_t *ht = (uint8_t *)malloc(th*tw*4);
    memset(ht, 0, 4*th*tw); // black bg


    for(rho = 0; rho < th; rho++)
    {
        for(theta = 0; theta < tw/*720*/; theta++)
        {
            double C = cos(RAD(theta));
            double S = sin(RAD(theta));
            uint32_t totalred = 0;
            uint32_t totalgreen = 0;
            uint32_t totalblue = 0;
            uint32_t totalpix = 0;
            if ( theta < 45 || (theta > 135 && theta < 225) || theta > 315) {
                for(y = 0; y < H; y++) {
                    double dx = W/2.0 + (rho - (H/2.0-y)*S)/C;
                    if ( dx < 0 || dx >= W ) continue;
                    x = floor(dx+.5);
                    if (x == W) continue;
                    totalpix++;
                    totalred += GR(x, y);
                    totalgreen += GG(x, y);
                    totalblue += GB(x, y);
                }
            } else {
                for(x = 0; x < W; x++) {
                    double dy = H/2.0 - (rho - (x - W/2.0)*C)/S;
                    if ( dy < 0 || dy >= H ) continue;
                    y = floor(dy+.5);
                    if (y == H) continue;
                    totalpix++;
                    totalred += GR(x, y);
                    totalgreen += GG(x, y);
                    totalblue += GB(x, y);      
                }
            }
            if ( totalpix > 0 ) {
                double dp = totalpix;
                SR(theta, rho) = (int)(totalred/dp)   &0xff;
                SG(theta, rho) = (int)(totalgreen/dp) &0xff;
                SB(theta, rho) = (int)(totalblue/dp)  &0xff;
            }
        }
    }
 
    *h = th;   // sqrt(W*W+H*H)/2
    *w = tw;   // 360
    *s = 4*tw;
    return ht;
}
 
int main(int argc, char **argv)
{
    cairo_surface_t *inputimg = NULL;
    cairo_surface_t *houghimg = NULL;

    uint8_t *houghdata = NULL, *inputdata = NULL;
    int w, h, s, bpp, format;

#if (CAIRO_HAS_PNG_FUNCTIONS==1)
    printf("cairo supports PNG\n");
#else
    printf("cairo does not support PNG\n");
#endif

    if ( argc < 3 ) return EXIT_FAILURE;

    printf("input file: %s\n", argv[1]);
    printf("output file: %s\n", argv[2]);

    inputimg = cairo_image_surface_create_from_png(argv[1]);

    printf("After create from png: %s\n",
        cairo_status_to_string(cairo_surface_status(inputimg)));

    w = cairo_image_surface_get_width(inputimg);
    h = cairo_image_surface_get_height(inputimg);
    s = cairo_image_surface_get_stride(inputimg);  
    format = cairo_image_surface_get_format(inputimg);
    switch(format)
    {
        case CAIRO_FORMAT_ARGB32: bpp = 4; break;
        case CAIRO_FORMAT_RGB24:  bpp = 3; break;
        case CAIRO_FORMAT_A8:     bpp = 1; break;
        default:
            fprintf(stderr, "unsupported %i\n", format);
            goto destroy;
    }

    inputdata = cairo_image_surface_get_data(inputimg);
    houghdata = houghtransform(inputdata, &w, &h, &s, bpp);

    printf("w=%d, h=%d\n", w, h);
    houghimg = cairo_image_surface_create_for_data(houghdata,
                        CAIRO_FORMAT_RGB24,
                        w, h, s);
    cairo_surface_write_to_png(houghimg, argv[2]);
 
destroy:
    if (inputimg != NULL) cairo_surface_destroy(inputimg);
    if (houghimg != NULL) cairo_surface_destroy(houghimg);

    return EXIT_SUCCESS;
}
