#include "hip/hip_runtime.h"
// Imported from https://rosettacode.org/wiki/Example:Hough_transform/C
// It will be used as a baseline to observe transformation
// Modified and Parallelized with CUDA by Vipin Bakshi and Andre Lo.

// GPU v?
// DETAILS: can take an extra param: ./<bin name> <input file> <output file> <grid size>
// if grid size is not entered, default is 12, which is default for v1

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <pthread.h>

#include "cairo.h"
#include "apptime.h"

#ifndef M_PI
#define M_PI 3.1415927
#endif

// These are macros to access the R, G and B values
// of the input  (d) data/ output data (ht) image buffers
#define GR(X,Y) (d[(stride)*(Y)+bpp*(X)+((2)%bpp)])
#define GG(X,Y) (d[(stride)*(Y)+bpp*(X)+((1)%bpp)])
#define GB(X,Y) (d[(stride)*(Y)+bpp*(X)+((0)%bpp)])
#define SR(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+2])
#define SG(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+1])
#define SB(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+0])
#define RAD(A)  (M_PI*((double)(A))/180.0)
#define tw       360

// Kernel
// todo: experiment with 3D instead of 1D grid?
static int grid;
__global__ void computationalkernel(uint8_t *d, uint8_t *ht, int W, int H, int stride, int bpp, int th)
{
    int rho, y, x;
    int theta = threadIdx.x + blockIdx.x * blockDim.x; // theta is based on grid/ block id

    for(rho = 0; rho < th; rho++)
    {
        double C = cos(RAD(theta));  // todo: call sincos instead?
        double S = sin(RAD(theta));
        uint32_t totalred = 0;
        uint32_t totalgreen = 0;
        uint32_t totalblue = 0;
        uint32_t totalpix = 0;
        if ( theta < 45 || (theta > 135 && theta < 225) || theta > 315) {
            for(y = 0; y < H; y++) {
                double dx = W/2.0 + (rho - (H/2.0-y)*S)/C;
                if ( dx < 0 || dx >= W ) continue;
                x = floor(dx+.5);
                if (x == W) continue;
                totalpix++;
                totalred += GR(x, y);
                totalgreen += GG(x, y);
                totalblue += GB(x, y);
            }
        } else {
            for(x = 0; x < W; x++) {
                double dy = H/2.0 - (rho - (x - W/2.0)*C)/S;
                if ( dy < 0 || dy >= H ) continue;
                y = floor(dy+.5);
                if (y == H) continue;
                totalpix++;
                totalred += GR(x, y);
                totalgreen += GG(x, y);
                totalblue += GB(x, y);
            }
        }
        if ( totalpix > 0 ) {
            double dp = totalpix;
            SR(theta, rho) = (int)(totalred/dp)   &0xff;
            SG(theta, rho) = (int)(totalgreen/dp) &0xff;
            SB(theta, rho) = (int)(totalblue/dp)  &0xff;
        }
    }
}


// d  is pointer to input data
// w, h, s is input data's width, height, and stridge
// bpp is bits per pixel of input data
uint8_t *houghtransform(uint8_t *h_in, int *w, int *h, int *s, int bpp)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    uint64_t measurement_time = 0;

    int W = *w, H = *h;
    int th = sqrt(W*W + H*H)/2.0;
    int outputBytes= th*tw*4;

    // alloc space for output buffer CPU side
    uint8_t *h_ht = (uint8_t *)malloc(outputBytes);

    apptime_start_session(&measurement_time);
    
    // alloc space for output buffer device side
    uint8_t *d_out;
    err = hipMalloc((void **)&d_out, outputBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate %d bytes for  d_out (error code %s)!\n", outputBytes, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemset((void *)d_out, 0, outputBytes); // black bg
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to hipMemset d_out (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("allocated output buffers\n");
    
    // alloc space and init input buffer device side
    uint8_t *d_in;
    err = hipMalloc((void **)&d_in, (*s * *h)); // bytes = stride * height
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_in (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_in, h_in, (*s * *h), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy d_in from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("allocated input buffers\n");
    measurement_time = apptime_stop_session(&measurement_time);
    printf("Allocations/ copy to device completed. Time:  %llu ns\n", measurement_time);

    apptime_start_session(&measurement_time);
    // todo: play with grid, block dimensions
    // right now this spawns 360 total kernels, for 360 values of theta
    computationalkernel <<<grid, (360/ grid)>>> (d_in, d_out, W, H, *s, bpp, th);

    hipDeviceSynchronize(); // wait for all GPU threads to complete
    printf("hipDeviceSynchronize done\n");

    measurement_time = apptime_stop_session(&measurement_time);
    printf("CUDA computations completed. Time:  %llu ns\n", measurement_time);


    apptime_start_session(&measurement_time);    
    // Copy resulting output from device
    hipMemcpy(h_ht, d_out, outputBytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy d_out from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("copy result back to host done\n");

    // Clean up
    err = hipFree(d_in);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_in (error code %s)!\n", hipGetErrorString(err));
    }

    err = hipFree(d_out);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_out (error code %s)!\n", hipGetErrorString(err));
    }
    
    measurement_time = apptime_stop_session(&measurement_time);
    printf("Copy back to host/ cleanup completed. Time:  %llu ns\n", measurement_time);
    
    // h, w, and s are returned as the height, width, stride of the output image
    // ht is the buffer containing the transformed output image
    *h = th;   // sqrt(W*W+H*H)/2
    *w = tw;   // 360
    *s = 4*tw; // 4 because 4 bytes per pixel output format
    return h_ht;
}

int main(int argc, char **argv)
{
    cairo_surface_t *inputimg = NULL;
    cairo_surface_t *houghimg = NULL;

    uint8_t *houghdata = NULL, *inputdata = NULL;
    int w, h, s, bpp, format;
    uint64_t measurement_time = 0;


#if (CAIRO_HAS_PNG_FUNCTIONS==1)
    printf("cairo supports PNG\n");
#else
    printf("cairo does not support PNG\n");
#endif

    if ( argc < 3 ) return EXIT_FAILURE;

    printf("input file: %s\n", argv[1]);
    printf("output file: %s\n", argv[2]);

    //todo: take in argv[3] as grid size?
    grid = 12; // must be a factor of 360 (we calculate using theta for every degree of 360 degs)

    if (argc > 3)
    {
        grid = atoi(argv[3]);
    }
    printf("grid = %d\n", grid);
    
    apptime_print_res();

    // Lets measure initialization time.
    apptime_start_session(&measurement_time);
    printf("Initialization...\n");
    inputimg = cairo_image_surface_create_from_png(argv[1]);

    printf("After create from png: %s\n",
        cairo_status_to_string(cairo_surface_status(inputimg)));

    w = cairo_image_surface_get_width(inputimg);
    h = cairo_image_surface_get_height(inputimg);
    s = cairo_image_surface_get_stride(inputimg);
    format = cairo_image_surface_get_format(inputimg);
    switch(format)
    {
        case CAIRO_FORMAT_ARGB32: bpp = 4; break;
        case CAIRO_FORMAT_RGB24:  bpp = 3; break;
        case CAIRO_FORMAT_A8:     bpp = 1; break;
        default:
            fprintf(stderr, "unsupported %i\n", format);
            goto destroy;
    }

    inputdata = cairo_image_surface_get_data(inputimg);
    measurement_time = apptime_stop_session(&measurement_time);
    printf("Initialization Completed. Time: %lld ns\n", measurement_time);

    printf("input buffer  width %d, height %d, stride %d, bpp %d\n",
        w, h, s, bpp);

    // Now lets measure the Hough Time.
    printf("Hough Transform using CUDA started...\n");
    //apptime_start_session(&measurement_time);

    houghdata = houghtransform(inputdata, &w, &h, &s, bpp);

    //measurement_time = apptime_stop_session(&measurement_time);
    //printf("Hought transform completed. Time:  %llu ns\n", measurement_time);

    printf("w=%d, h=%d\n", w, h);
    houghimg = cairo_image_surface_create_for_data(houghdata,
                        CAIRO_FORMAT_RGB24,
                        w, h, s);
    cairo_surface_write_to_png(houghimg, argv[2]);

destroy:
    if (inputimg != NULL) cairo_surface_destroy(inputimg);
    if (houghimg != NULL) cairo_surface_destroy(houghimg);

    return EXIT_SUCCESS;
}
