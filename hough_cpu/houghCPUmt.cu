#include "hip/hip_runtime.h"
// Imported from https://rosettacode.org/wiki/Example:Hough_transform/C
// It will be used as a baseline to observe transformation
// Modified and Parallelized by Vipin Bakshi and Andre Lo.

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <pthread.h>
 
#include "cairo.h"
#include "apptime.h"
 
#ifndef M_PI
#define M_PI 3.1415927
#endif
 
#define GR(X,Y) (d[(*s)*(Y)+bpp*(X)+((2)%bpp)])
#define GG(X,Y) (d[(*s)*(Y)+bpp*(X)+((1)%bpp)])
#define GB(X,Y) (d[(*s)*(Y)+bpp*(X)+((0)%bpp)])
#define SR(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+2])
#define SG(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+1])
#define SB(X,Y) (ht[4*tw*((Y)%th)+4*((X)%tw)+0])
#define RAD(A)  (M_PI*((double)(A))/180.0)

#define PTHREAD_MAX_THREADS        8

struct computationblock_limits
{
  int rho_start;
  int rho_end;
  int tid;
};

// PTHREAD SUPPORT
static pthread_t threads[PTHREAD_MAX_THREADS];
pthread_attr_t attr;

// Global support.
static int  W, H, th, tw;
static uint8_t* ht;
static uint8_t* d;
static int* s;
static int bpp;

static struct computationblock_limits cl[PTHREAD_MAX_THREADS];

// The worker thread.
void * computationblock(void* data);


uint8_t *houghtransform(uint8_t *dd, int *w, int *h, int *ss, int bbpp)
{
    int ii;
    W = *w, H = *h;
    th = sqrt(W*W + H*H)/2.0;
    tw = 360;
    ht = (uint8_t *)malloc(th*tw*4);
    memset(ht, 0, 4*th*tw); // black bg
    memset(&cl, 0, sizeof(cl));
    d = dd;
    s = ss;
    bpp = bbpp;
    
    // Create pthread attribute with JOINABLE property.
    pthread_attr_init(&attr);
    pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);

    // Disptach threads for work.
    for (ii = 0; ii < PTHREAD_MAX_THREADS; ii++)
    {
      int rc;

      // Load structure with points info to send to thread.
      cl[ii].rho_start = (th / PTHREAD_MAX_THREADS) * ii;
      cl[ii].tid = ii;

      // Check if this is the last thread to be dispatched 
      //	and we have a residue of points to be computed.
      //	If it is true, add the residue points as load to the last thread.
      if ((ii == PTHREAD_MAX_THREADS - 1) &&
          (th % PTHREAD_MAX_THREADS))
      {
	// Last thread to dispatch. Add remaining points to compute to this thread.
	cl[ii].rho_end = (th / PTHREAD_MAX_THREADS)
	                   + cl[ii].rho_start
	                   + (th % PTHREAD_MAX_THREADS);
      }
      else
      {
	// Not the last thread or (last thread but no residue in the number of points to be computed.
	cl[ii].rho_end = (th / PTHREAD_MAX_THREADS)
	  + cl[ii].rho_start;
      }

      rc = pthread_create(&threads[ii], &attr, computationblock, (void *) &cl[ii]);
      if (rc)
      {
	// Error creating threads.
	printf("ERROR! Return code from pthread_create() is %d\n", rc);
	exit(-1);
       }
    }

    // BARRIER for Computation.
    ii = 0;
    for (ii = 0; ii < PTHREAD_MAX_THREADS; ii++)
    {
      int rc;
      void* status;

      rc = pthread_join(threads[ii], &status);
      if (rc)
      {
	printf("ERROR! Return code from pthread_join() is %d\n.", rc);
	exit(-1);
      }
    }
	
    *h = th;   // sqrt(W*W+H*H)/2
    *w = tw;   // 360
    *s = 4*tw;

    pthread_attr_destroy(&attr);

    return ht;
}

void * computationblock(void* data)
{
   uint64_t thread_time;
   int rho, theta, y, x;
   struct computationblock_limits * cl  = (struct computationblock_limits*)(data);

   apptime_start_session(&thread_time);
   for(rho = cl->rho_start; rho < cl->rho_end; rho++)
   {
     for(theta = 0; theta < tw/*720*/; theta++)
       {
	 double C = cos(RAD(theta));
	 double S = sin(RAD(theta));
	 uint32_t totalred = 0;
	 uint32_t totalgreen = 0;
	 uint32_t totalblue = 0;
	 uint32_t totalpix = 0;
	 if ( theta < 45 || (theta > 135 && theta < 225) || theta > 315) {
	   for(y = 0; y < H; y++) {
	     double dx = W/2.0 + (rho - (H/2.0-y)*S)/C;
	     if ( dx < 0 || dx >= W ) continue;
	     x = floor(dx+.5);
	     if (x == W) continue;
	     totalpix++;
	     totalred += GR(x, y);
	     totalgreen += GG(x, y);
	     totalblue += GB(x, y);
	   }
	 } else {
	   for(x = 0; x < W; x++) {
	     double dy = H/2.0 - (rho - (x - W/2.0)*C)/S;
	     if ( dy < 0 || dy >= H ) continue;
	     y = floor(dy+.5);
	     if (y == H) continue;
	     totalpix++;
	     totalred += GR(x, y);
	     totalgreen += GG(x, y);
	     totalblue += GB(x, y);      
	   }
	 }
	 if ( totalpix > 0 ) {
	   double dp = totalpix;
	   SR(theta, rho) = (int)(totalred/dp)   &0xff;
	   SG(theta, rho) = (int)(totalgreen/dp) &0xff;
	   SB(theta, rho) = (int)(totalblue/dp)  &0xff;
	 }
       }
    }
    
    thread_time = apptime_stop_session(&thread_time);
    printf("Thread %d exited. Time: %lld nm\n", cl->tid, thread_time);

    pthread_exit((void*) &cl->tid);
}
 
int main(int argc, char **argv)
{
    cairo_surface_t *inputimg = NULL;
    cairo_surface_t *houghimg = NULL;

    uint8_t *houghdata = NULL, *inputdata = NULL;
    int w, h, s, bpp, format;
    uint64_t measurement_time = 0;
    

#if (CAIRO_HAS_PNG_FUNCTIONS==1)
    printf("cairo supports PNG\n");
#else
    printf("cairo does not support PNG\n");
#endif

    if ( argc < 3 ) return EXIT_FAILURE;

    printf("input file: %s\n", argv[1]);
    printf("output file: %s\n", argv[2]);

    apptime_print_res();

    // Lets measure initialization time.
    apptime_start_session(&measurement_time);
    printf("Initialization...\n");
    inputimg = cairo_image_surface_create_from_png(argv[1]);

    printf("After create from png: %s\n",
        cairo_status_to_string(cairo_surface_status(inputimg)));

    w = cairo_image_surface_get_width(inputimg);
    h = cairo_image_surface_get_height(inputimg);
    s = cairo_image_surface_get_stride(inputimg);  
    format = cairo_image_surface_get_format(inputimg);
    switch(format)
    {
        case CAIRO_FORMAT_ARGB32: bpp = 4; break;
        case CAIRO_FORMAT_RGB24:  bpp = 3; break;
        case CAIRO_FORMAT_A8:     bpp = 1; break;
        default:
            fprintf(stderr, "unsupported %i\n", format);
            goto destroy;
    }

    inputdata = cairo_image_surface_get_data(inputimg);
    measurement_time = apptime_stop_session(&measurement_time);
    printf("Initialization Completed. Time: %lld ns\n", measurement_time);

    // Now lets measure the Hough Time.
    printf("Hough Transform started...\n");
    apptime_start_session(&measurement_time);
    
    houghdata = houghtransform(inputdata, &w, &h, &s, bpp);
    
    measurement_time = apptime_stop_session(&measurement_time);
    printf("Hought transform completed. Time:  %llu ns\n", measurement_time);
    
    printf("w=%d, h=%d\n", w, h);
    houghimg = cairo_image_surface_create_for_data(houghdata,
                        CAIRO_FORMAT_RGB24,
                        w, h, s);
    cairo_surface_write_to_png(houghimg, argv[2]);
 
destroy:
    if (inputimg != NULL) cairo_surface_destroy(inputimg);
    if (houghimg != NULL) cairo_surface_destroy(houghimg);
    pthread_exit(NULL);
    
    return EXIT_SUCCESS;
}
